#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#include "../timer.h"
#include "../funcs.h"

#define ROW_SIZE	360
#define	COLUMN_SIZE	360

#define REPEAT_TIMES 	20
#define MAX_TIMES	500	
#define TERMCHECK_TIMES 10

#define THREAD_SIZE	512

//#define TERMINATION_CHECK

__global__ void Game_of_Life_Kernel(int* cells, int * np_cells, int * size, int * columns) {
	int cell_index = threadIdx.x + blockIdx.x * blockDim.x;

	if( cell_index < *size )
	{ 
		/*find neighbours*/
		int x = cell_index % *columns;
		int y = cell_index - x;
		int Left  = (x + *(columns) - 1) % *columns;
		int Right = (x + 1) % *columns;
		int Up = (y + *size - *columns) % *size;
		int Down = (y + *columns) % *size;
		 
		int neighbours =  cells[Left + Up]     /*north west*/
				+ cells[x + Up]        /*north*/
				+ cells[Right + Up]    /*north east*/
				+ cells[Left + y]      /*west*/
				+ cells[Right + y]     /*east*/
				+ cells[Left + Down]   /*south west*/
				+ cells[x + Down]      /*south*/
				+ cells[Right + Down]; /*south east*/

		if( (neighbours == 3) || ( (cells[cell_index] ==  ALIVE) && (neighbours == 2) ) )
			np_cells[cell_index] = ALIVE;
		else
			np_cells[cell_index] = DEAD;
	}
}

/*---------------->*/
int main() {

	int * cells 	 = NULL;
	int * np_cells = NULL;
	int * temp = NULL;
	int array_size;
	int	array_columns;
	int	size;
	int i,j;
	int n = 0;
	double loop_start, loop_finish;
	double start, finish;

#ifdef TERMINATION_CHECK
	int	not_dead=0;
	int	not_duplicate=0;
#endif
	int * dcells = NULL;
	int * dnp_cells = NULL;
	int * darray_size = NULL;
	int * darray_columns = NULL;

	srand(time(NULL));
	size = ROW_SIZE * COLUMN_SIZE * sizeof(int);

	/*malloc the cell arrays*/
	if( (cells = (int*) malloc(size)) == NULL )
		exit(EXIT_FAILURE);

	if( (np_cells = (int*) malloc(size)) == NULL )
		exit(EXIT_FAILURE);

	/*how many threads in a block*/
	array_size = ROW_SIZE * COLUMN_SIZE;
	array_columns = COLUMN_SIZE;

	/*allocate device space*/
	hipMalloc((void**)&dcells, size);
	hipMalloc((void**)&dnp_cells, size);

	hipMalloc((void**)&darray_size, sizeof(int));
	hipMalloc((void**)&darray_columns, sizeof(int));

	/*setup input values*/
	for(i=0; i<ROW_SIZE; i++)
	{
		for(j=0; j<COLUMN_SIZE; j++)
		{
			cells[COLUMN_SIZE*i +j] = rand() % 2;
			np_cells[COLUMN_SIZE*i + j] = 0;
		}
	}

	/*================================================> start overall calculation time*/
	GET_TIME(start);
	while( n < MAX_TIMES)
	{
		/*================================================> start loop calculation time*/
		GET_TIME(loop_start);

		/*copy inputs to device*/
		hipMemcpy(dcells, cells, size, hipMemcpyHostToDevice);
		hipMemcpy(dnp_cells, np_cells, size, hipMemcpyHostToDevice);

		hipMemcpy(darray_size, &array_size, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(darray_columns, &array_columns, sizeof(int), hipMemcpyHostToDevice);

		/*call kernel*/
		Game_of_Life_Kernel<<<array_size/THREAD_SIZE, THREAD_SIZE>>>(dcells, dnp_cells, darray_size, darray_columns);

		/*copy result to host*/
		hipMemcpy(cells, dcells, size, hipMemcpyDeviceToHost);
		hipMemcpy(np_cells, dnp_cells, size, hipMemcpyDeviceToHost);

#ifdef TERMINATION_CHECK
		if( n % TERMCHECK_TIMES == 0)	
		{	
			not_duplicate = 0;
			not_dead = 0;

			/*compare current phase with the next one && check if everything is dead*/
			for(i=0; i<ROW_SIZE; i++) {
				for(j=0; j<COLUMN_SIZE; j++) {
					if( cells[COLUMN_SIZE*i + j] != np_cells[COLUMN_SIZE*i + j] )
						not_duplicate = 1;
				}
			}
			
			for(i=0; i<ROW_SIZE; i++) {
				for(j=0; j<COLUMN_SIZE; j++) {
					if( cells[COLUMN_SIZE*i + j] == ALIVE )
						not_dead = 1;
				}
			}

			if( not_dead == 0 )
				printf("!--->Every cell is dead, program about to exit.\n");

			if( not_duplicate == 0 )
				printf("!--->Current cell generation is the same as the next one.\n");

			if( not_dead == 0  || not_duplicate == 0) {
				/*free memory*/
				free(cells);
				free(np_cells);

				cells = NULL;
				np_cells = NULL;

				hipFree(dcells);
				hipFree(dnp_cells);
				hipFree(darray_size);
				hipFree(darray_columns);

				dcells = NULL;
				dnp_cells = NULL;
				darray_size = NULL;
				darray_columns = NULL;

				exit(EXIT_SUCCESS);
			}
		}
#endif
		/*================================================> finish loop calculation time*/
		GET_TIME(loop_finish);
		if(n % REPEAT_TIMES == 0)
			printf("->Elapsed time = %.10f seconds\n", loop_finish-loop_start);

		/*swap arrays*/
		temp = cells;
		cells = np_cells;
		np_cells = temp;

		/*increment loop counter*/
		n++;
	}

	/*================================================> finish overall calculation time*/
	GET_TIME(finish);
	printf("->>Elapsed overall time = %.10f seconds\n", finish-start);

	/*free memory*/
	free(cells);
	free(np_cells);

	cells = NULL;
	np_cells = NULL;

	hipFree(dcells);
	hipFree(dnp_cells);
	hipFree(darray_size);
	hipFree(darray_columns);

	dcells = NULL;
	dnp_cells = NULL;
	darray_size = NULL;
	darray_columns = NULL;

	exit(EXIT_SUCCESS);
}
